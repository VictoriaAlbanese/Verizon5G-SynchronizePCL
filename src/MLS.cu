#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////
//
// Programmer: Victoria Albanese
// Filename: MLS.cu
//
// Purpose: Cuda MLS stuff, from this source
// https://github.com/colincsl/Kinect-Projects/blob/master/MovingLeastSquares_PCL/MLS.h
//
////////////////////////////////////////////////////////////////

#include "MLS.h"

void thrustPCL_AOS(boost::shared_ptr<PointCloudAOS<Device> > cloud,
              PointCloudAOS<Device>::Ptr out,
              int nn_connectivity,
              float smoothness)
{
    const int size = 640*480;
    device_vector<int> indices(size);
    thrust::sequence(indices.begin(), indices.end());

    MovingLeastSquares kernel = MovingLeastSquares(cloud, nn_connectivity, smoothness);

    thrust::transform(make_zip_iterator(make_tuple(cloud->points.begin(), indices.begin())),
                      make_zip_iterator(make_tuple(cloud->points.end(), indices.end())), 
                      out->points.begin(),
                      kernel);
}

template <typename Tuple>
__host__ __device__
PointXYZRGB MovingLeastSquares::operator()(const Tuple &data) const
{
    PointXYZRGB point = thrust::get<0>(data);
    int index = thrust::get<1>(data);

    PointXYZRGB point2, pOut, nn_;

    const int nn_count = 49;
    int stride[] =  {-1923, -1922, -1921, -1920, -1919, -1918, -1917,\
                    -1283, -1282, -1281, -1280, -1279, -1278, 1977,\
                    -641, -642, -641, -640, -639,-638,-637, \
                    -3, -2, -1, 0, 1, 2, 3,\
                     638, 639, 640, 641, 642, 643,\
                     1277, 1278, 1279, 1280, 1281, 1282, 1283, \
                     1917, 1918, 1919, 1920, 1921, 1922, 1923};

    float3 centroid = make_float3(0.0,0.0,0.0);
    float3 neighbors[nn_count];

    // Find centroid --------------------------------------------

    int current_ind=0;
    int ind_count=0;
    int max_ind = 640*480;
    //float thresh = .01*nn_connectivity;
    
    for (int i=0; i<nn_count; i++)
    {
        current_ind = index+stride[i];
        if (current_ind >= 0 && current_ind < max_ind)
        {
            nn_ = points[current_ind];

            if (nn_.x > 0.0);
            {
                centroid.x += nn_.x;
                centroid.y += nn_.y;
                centroid.z += nn_.z;

                neighbors[ind_count] = make_float3(nn_.x, nn_.y, nn_.z);
                ind_count++;
            }
        }
    }

    if (ind_count < 3) return pOut;

    centroid.x /= (float)ind_count;
    centroid.y /= (float)ind_count;
    centroid.z /= (float)ind_count;            
        
    // Calculate covariance -----------------------------------

    CovarianceMatrix cov;
    cov.data[0] = make_float3(0,0,0);
    cov.data[1] = make_float3(0,0,0);
    cov.data[2] = make_float3(0,0,0);

    float3 tmp_nn;
    for (int k=0; k<ind_count; k++)
    {
        tmp_nn = neighbors[k];
        cov.data[0].x += (tmp_nn.x-centroid.x)*(tmp_nn.x-centroid.x);
        cov.data[0].y += (tmp_nn.x-centroid.x)*(tmp_nn.y-centroid.y);
        cov.data[0].z += (tmp_nn.x-centroid.x)*(tmp_nn.z-centroid.z);
        cov.data[1].y += (tmp_nn.y-centroid.y)*(tmp_nn.y-centroid.y);
        cov.data[1].z += (tmp_nn.y-centroid.y)*(tmp_nn.z-centroid.z);
        cov.data[2].z += (tmp_nn.z-centroid.z)*(tmp_nn.z-centroid.z);
    }
        
    cov.data[0].x /= (ind_count-1);
    cov.data[0].y /= (ind_count-1);
    cov.data[0].z /= (ind_count-1);
    cov.data[1].y /= (ind_count-1);
    cov.data[1].z /= (ind_count-1);
    cov.data[2].z /= (ind_count-1);

    // fill in the lower triangle (symmetry)
    cov.data[1].x = cov.data[0].y;
    cov.data[2].x = cov.data[0].z;
    cov.data[2].y = cov.data[1].z;

    // Eigen Stuff? -----------------------------------

    CovarianceMatrix evecs;
    float3 evals;

    pcl::cuda::eigen33 (cov, evecs, evals);

    float3 pointC;
    float3 normal = evecs.data[0];
    float eigenvalue = evals.z;

    pointC.x = point.x; 
    pointC.y = point.y; 
    pointC.z = point.z;
        
    float model_coeff = -1*(normal.x*centroid.x + normal.y*centroid.y + normal.z*centroid.z);
    float distance = pointC.x*normal.x + pointC.y*normal.y + pointC.z*normal.z + model_coeff;
    pointC -= distance * normal;

    float curvature = cov.data[0].x+cov.data[1].y+cov.data[2].z; // curv = tr(covariance)
    if (curvature != 0) curvature = fabs(eigenvalue / curvature);

    float nn_dist[nn_count];
    for (int i=0; i<ind_count; i++)
    {
        neighbors[i] -= pointC;
        nn_dist[i] = neighbors[i].x*neighbors[i].x + neighbors[i].y*neighbors[i].y + neighbors[i].z*neighbors[i].z;
    }

    // Init polynomial params - assume 3 coeffs for now
    const int nr_coeff = 3; // number of coeffs in polynomial
    const float sqr_gauss_param = smoothness;

    float weight_vec[nn_count];
    float f_vec[nn_count];
    float3 c_vec;
    float3 P[nn_count];
    float3 P_weight[nn_count];

    //Local coordinate system
    float3 v = unitOrthogonal(normal);
    float3 u = cross(normal, v);

    float u_coord, v_coord, u_pow, v_pow;
    for (int i=0; i<ind_count; i++)
    {
        // Compute weight
        weight_vec[i] = exp(-nn_dist[i] / sqr_gauss_param);

        // Transform coords
        u_coord = neighbors[i].x*u.x + neighbors[i].y*u.y +  neighbors[i].z*u.z;
        v_coord = neighbors[i].x*v.x + neighbors[i].y*v.y +  neighbors[i].z*v.z;
        f_vec[i]= dot(neighbors[i], normal);

        u_pow = 1;
        for(int i2=0; i2<nr_coeff-1; i2++)
        {
            v_pow=1;
            P[i].x = u_pow*v_pow;
            v_pow *= v_coord;
            P[i].y = u_pow*v_pow;
            v_pow *= v_coord;
            P[i].z = u_pow*v_pow;
            v_pow *= v_coord;

            u_pow *= u_coord;
        }
    }

    // P is NNx3
    // P_weight is NNx3
    // P_weight_Pt is 3x3
    for (int i=0; i<ind_count; i++)
    {
        P_weight[i].x = P[i].x*weight_vec[i];
        P_weight[i].x = P[i].x*weight_vec[i];
        P_weight[i].x = P[i].x*weight_vec[i];
    }

    c_vec.x=0.0; c_vec.y=0.0; c_vec.z=0.0;
    for (int i=0; i<ind_count; i++)
    {
        c_vec.x += P_weight[i].x*f_vec[i];
        c_vec.y += P_weight[i].y*f_vec[i];
        c_vec.z += P_weight[i].z*f_vec[i];
    }

    pointC += c_vec.x*normal;

    pOut.x = pointC.x;
    pOut.y = pointC.y;
    pOut.z = pointC.z;
    pOut.rgb.b = 120;
    pOut.rgb.r = 120;

    return pOut;
};

////////////////////////////////////////////////////////////////
